#include "hip/hip_runtime.h"
/*!
 * \file cuda_multicorrelator.cu
 * \brief High optimized CUDA GPU vector multiTAP correlator class
 * \authors <ul>
 *          <li> Javier Arribas, 2015. jarribas(at)cttc.es
 *          </ul>
 *
 * Class that implements a high optimized vector multiTAP correlator class for NVIDIA CUDA GPUs
 *
 * -------------------------------------------------------------------------
 *
 * Copyright (C) 2010-2015  (see AUTHORS file for a list of contributors)
 *
 * GNSS-SDR is a software defined Global Navigation
 *          Satellite Systems receiver
 *
 * This file is part of GNSS-SDR.
 *
 * GNSS-SDR is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * GNSS-SDR is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with GNSS-SDR. If not, see <http://www.gnu.org/licenses/>.
 *
 * -------------------------------------------------------------------------
 */

///////////////////////////////////////////////////////////////////////////////
// On G80-class hardware 24-bit multiplication takes 4 clocks per warp
// (the same as for floating point  multiplication and addition),
// whereas full 32-bit multiplication takes 16 clocks per warp.
// So if integer multiplication operands are  guaranteed to fit into 24 bits
// (always lie withtin [-8M, 8M - 1] range in signed case),
// explicit 24-bit multiplication is preferred for performance.
///////////////////////////////////////////////////////////////////////////////
#define IMUL(a, b) __mul24(a, b)

#include "cuda_multicorrelator.h"

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#define ACCUM_N 1024

///////////////////////////////////////////////////////////////////////////////
// Calculate scalar products of VectorN vectors of ElementN elements on GPU
// Parameters restrictions:
// 1) ElementN is strongly preferred to be a multiple of warp size to
//    meet alignment constraints of memory coalescing.
// 2) ACCUM_N must be a power of two.
///////////////////////////////////////////////////////////////////////////////


__global__ void scalarProdGPUCPXxN_shifts(
    GPU_Complex *d_corr_out,
    GPU_Complex *d_sig_in,
    GPU_Complex *d_local_codes_in,
    int *d_shifts_samples,
    int vectorN,
    int elementN
)
{
    //Accumulators cache
    __shared__ GPU_Complex accumResult[ACCUM_N];

    ////////////////////////////////////////////////////////////////////////////
    // Cycle through every pair of vectors,
    // taking into account that vector counts can be different
    // from total number of thread blocks
    ////////////////////////////////////////////////////////////////////////////
    for (int vec = blockIdx.x; vec < vectorN; vec += gridDim.x)
    {
        int vectorBase = IMUL(elementN, vec);
        int vectorEnd  = vectorBase + elementN;

        ////////////////////////////////////////////////////////////////////////
        // Each accumulator cycles through vectors with
        // stride equal to number of total number of accumulators ACCUM_N
        // At this stage ACCUM_N is only preferred be a multiple of warp size
        // to meet memory coalescing alignment constraints.
        ////////////////////////////////////////////////////////////////////////
        for (int iAccum = threadIdx.x; iAccum < ACCUM_N; iAccum += blockDim.x)
        {
        	GPU_Complex sum = GPU_Complex(0,0);

            for (int pos = vectorBase + iAccum; pos < vectorEnd; pos += ACCUM_N)
            {
                //sum = sum + d_sig_in[pos-vectorBase] * d_nco_in[pos-vectorBase] * d_local_codes_in[pos];
            	//sum = sum + d_sig_in[pos-vectorBase] * d_local_codes_in[pos];
            	sum.multiply_acc(d_sig_in[pos-vectorBase],d_local_codes_in[pos-vectorBase+d_shifts_samples[vec]]);
            }
            accumResult[iAccum] = sum;
        }

        ////////////////////////////////////////////////////////////////////////
        // Perform tree-like reduction of accumulators' results.
        // ACCUM_N has to be power of two at this stage
        ////////////////////////////////////////////////////////////////////////
        for (int stride = ACCUM_N / 2; stride > 0; stride >>= 1)
        {
            __syncthreads();

            for (int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x)
            {
                accumResult[iAccum] += accumResult[stride + iAccum];
            }
        }

        if (threadIdx.x == 0)
        	{
        		d_corr_out[vec] = accumResult[0];
        	}
    }
}


__global__ void scalarProdGPUCPXxN(
    GPU_Complex *d_corr_out,
    GPU_Complex *d_sig_in,
    GPU_Complex *d_local_codes_in,
    int vectorN,
    int elementN
)
{
    //Accumulators cache
    __shared__ GPU_Complex accumResult[ACCUM_N];

    ////////////////////////////////////////////////////////////////////////////
    // Cycle through every pair of vectors,
    // taking into account that vector counts can be different
    // from total number of thread blocks
    ////////////////////////////////////////////////////////////////////////////
    for (int vec = blockIdx.x; vec < vectorN; vec += gridDim.x)
    {
        int vectorBase = IMUL(elementN, vec);
        int vectorEnd  = vectorBase + elementN;

        ////////////////////////////////////////////////////////////////////////
        // Each accumulator cycles through vectors with
        // stride equal to number of total number of accumulators ACCUM_N
        // At this stage ACCUM_N is only preferred be a multiple of warp size
        // to meet memory coalescing alignment constraints.
        ////////////////////////////////////////////////////////////////////////
        for (int iAccum = threadIdx.x; iAccum < ACCUM_N; iAccum += blockDim.x)
        {
        	GPU_Complex sum = GPU_Complex(0,0);

            for (int pos = vectorBase + iAccum; pos < vectorEnd; pos += ACCUM_N)
            {
                //sum = sum + d_sig_in[pos-vectorBase] * d_nco_in[pos-vectorBase] * d_local_codes_in[pos];
            	//sum = sum + d_sig_in[pos-vectorBase] * d_local_codes_in[pos];
            	sum.multiply_acc(d_sig_in[pos-vectorBase],d_local_codes_in[pos]);
            }
            accumResult[iAccum] = sum;
        }

        ////////////////////////////////////////////////////////////////////////
        // Perform tree-like reduction of accumulators' results.
        // ACCUM_N has to be power of two at this stage
        ////////////////////////////////////////////////////////////////////////
        for (int stride = ACCUM_N / 2; stride > 0; stride >>= 1)
        {
            __syncthreads();

            for (int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x)
            {
                accumResult[iAccum] += accumResult[stride + iAccum];
            }
        }

        if (threadIdx.x == 0)
        	{
        		d_corr_out[vec] = accumResult[0];
        	}
    }
}


//*********** CUDA processing **************
// Treads: a minimal parallel execution code on GPU
// Blocks: a set of N threads
/**
 * CUDA Kernel Device code
 *
 * Computes the vectorial product of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void CUDA_32fc_x2_multiply_32fc(  GPU_Complex *A,   GPU_Complex  *B, GPU_Complex  *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] =  A[i] * B[i];
    }
}


/**
 * CUDA Kernel Device code
 *
 * Computes the carrier Doppler wipe-off by integrating the NCO in the CUDA kernel
 */
__global__ void
CUDA_32fc_Doppler_wipeoff(  GPU_Complex *sig_out, GPU_Complex *sig_in, float rem_carrier_phase_in_rad, float phase_step_rad, int numElements)
{
	//*** NCO CPU code (GNURadio FXP NCO)
	//float sin_f, cos_f;
	//float phase_step_rad = static_cast<float>(2 * GALILEO_PI) * d_carrier_doppler_hz / static_cast<float>(d_fs_in);
	//int phase_step_rad_i = gr::fxpt::float_to_fixed(phase_step_rad);
	//int phase_rad_i = gr::fxpt::float_to_fixed(d_rem_carr_phase_rad);
	//
	//for(int i = 0; i < d_current_prn_length_samples; i++)
	//    {
	//        gr::fxpt::sincos(phase_rad_i, &sin_f, &cos_f);
	//        d_carr_sign[i] = std::complex<float>(cos_f, -sin_f);
	//        phase_rad_i += phase_step_rad_i;
	//    }

	// CUDA version of floating point NCO and vector dot product integrated

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    float sin;
    float cos;
    if (i < numElements)
    {
    	__sincosf(rem_carrier_phase_in_rad + i*phase_step_rad, &sin, &cos);
    	sig_out[i] =  sig_in[i] * GPU_Complex(cos,-sin);
    }
}


/**
 * CUDA Kernel Device code
 *
 * Computes the vectorial product of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void
CUDA_32fc_x2_add_32fc(  GPU_Complex *A,   GPU_Complex  *B, GPU_Complex  *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] =  A[i] * B[i];
    }
}


bool cuda_multicorrelator::init_cuda(const int argc, const char **argv, int signal_length_samples, int local_codes_length_samples, int n_correlators)
{
	// use command-line specified CUDA device, otherwise use device with highest Gflops/s
//	findCudaDevice(argc, (const char **)argv);
      hipDeviceProp_t  prop;
    int num_devices, device;
    hipGetDeviceCount(&num_devices);
    num_gpu_devices=num_devices;
    if (num_devices > 1) {
          int max_multiprocessors = 0, max_device = 0;
          for (device = 0; device < num_devices; device++) {
                  hipDeviceProp_t properties;
                  hipGetDeviceProperties(&properties, device);
                  if (max_multiprocessors < properties.multiProcessorCount) {
                          max_multiprocessors = properties.multiProcessorCount;
                          max_device = device;
                  }
                  printf("Found GPU device # %i\n",device);
          }
		    //set random device!
		    selected_device=(rand() % num_devices);
		    printf("selected_device=%i\n",selected_device);
          hipGetDeviceProperties( &prop, selected_device );
          //debug code
          if (prop.canMapHostMemory != 1) {
              printf( "Device can not map memory.\n" );
          }
          printf("L2 Cache size= %u \n",prop.l2CacheSize);
          printf("maxThreadsPerBlock= %u \n",prop.maxThreadsPerBlock);
          printf("maxGridSize= %i \n",prop.maxGridSize[0]);
          printf("sharedMemPerBlock= %lu \n",prop.sharedMemPerBlock);
          printf("deviceOverlap= %i \n",prop.deviceOverlap);
  	    printf("multiProcessorCount= %i \n",prop.multiProcessorCount);
    }else{
    		selected_device=0;
    	    int whichDevice;
    	    hipGetDevice( &whichDevice );
    	    hipGetDeviceProperties( &prop, whichDevice );
    	    //debug code
    	    if (prop.canMapHostMemory != 1) {
    	        printf( "Device can not map memory.\n" );
    	    }

    	    printf("L2 Cache size= %u \n",prop.l2CacheSize);
    	    printf("maxThreadsPerBlock= %u \n",prop.maxThreadsPerBlock);
    	    printf("maxGridSize= %i \n",prop.maxGridSize[0]);
    	    printf("sharedMemPerBlock= %lu \n",prop.sharedMemPerBlock);
    	    printf("deviceOverlap= %i \n",prop.deviceOverlap);
    	    printf("multiProcessorCount= %i \n",prop.multiProcessorCount);
    }


	//checkCudaErrors(hipFuncSetCacheConfig(reinterpret_cast<const void*>(CUDA_32fc_x2_multiply_x2_dot_prod_32fc_), hipFuncCachePreferShared));


    // ALLOCATE GPU MEMORY FOR INPUT/OUTPUT and INTERNAL vectors

    size_t size = signal_length_samples * sizeof(GPU_Complex);
    hipSetDevice(selected_device); //generates a random number between 0 and num_devices to split the threads between GPUs
    
	checkCudaErrors(hipMalloc((void **)&d_sig_in, size));
	//checkCudaErrors(hipMalloc((void **)&d_nco_in, size));
	checkCudaErrors(hipMalloc((void **)&d_sig_doppler_wiped, size));

	// old version: all local codes are independent vectors
	//checkCudaErrors(hipMalloc((void **)&d_local_codes_in, size*n_correlators));

	// new version: only one vector with extra samples to shift the local code for the correlator set
	// Required: The last correlator tap in d_shifts_samples has the largest sample shift
    size_t size_local_code_bytes = local_codes_length_samples * sizeof(GPU_Complex);
	checkCudaErrors(hipMalloc((void **)&d_local_codes_in, size_local_code_bytes));
	checkCudaErrors(hipMalloc((void **)&d_shifts_samples, size+sizeof(int)*n_correlators));

	//scalars
	checkCudaErrors(hipMalloc((void **)&d_corr_out, sizeof(std::complex<float>)*n_correlators));

    // Launch the Vector Add CUDA Kernel
	threadsPerBlock = 256;
    blocksPerGrid =(int)(signal_length_samples+threadsPerBlock-1)/threadsPerBlock;

	hipStreamCreate (&stream1) ;
	hipStreamCreate (&stream2) ;
	return true;
}


bool cuda_multicorrelator::Carrier_wipeoff_multicorrelator_cuda(
		std::complex<float>* corr_out,
		const std::complex<float>* sig_in,
		const std::complex<float>* local_codes_in,
		float rem_carrier_phase_in_rad,
		float phase_step_rad,
		const int *shifts_samples,
		int signal_length_samples,
		int n_correlators)
	{

	size_t memSize = signal_length_samples * sizeof(std::complex<float>);
    hipSetDevice(selected_device); //generates a random number between 0 and num_devices to split the threads between GPUs
	// input signal CPU -> GPU copy memory

    checkCudaErrors(hipMemcpyAsync(d_sig_in, sig_in, memSize,
                                    hipMemcpyHostToDevice, stream1));

    //***** NOTICE: NCO is computed on-the-fly, not need to copy NCO into GPU! ****
    //checkCudaErrors(hipMemcpyAsync(d_nco_in, nco_in, memSize,
    //                                hipMemcpyHostToDevice, stream1));


	// old version: all local codes are independent vectors
    //checkCudaErrors(hipMemcpyAsync(d_local_codes_in, local_codes_in, memSize*n_correlators,
    //                                hipMemcpyHostToDevice, stream2));

	// new version: only one vector with extra samples to shift the local code for the correlator set
	// Required: The last correlator tap in d_shifts_samples has the largest sample shift

    // local code CPU -> GPU copy memory
    checkCudaErrors(hipMemcpyAsync(d_local_codes_in, local_codes_in, memSize+sizeof(std::complex<float>)*shifts_samples[n_correlators-1],
                                    hipMemcpyHostToDevice, stream2));
    // Correlator shifts vector CPU -> GPU copy memory
    checkCudaErrors(hipMemcpyAsync(d_shifts_samples, shifts_samples, sizeof(int)*n_correlators,
                                    hipMemcpyHostToDevice, stream2));


    //Launch carrier wipe-off kernel here, while local codes are being copied to GPU!
    checkCudaErrors(hipStreamSynchronize(stream1));
    CUDA_32fc_Doppler_wipeoff<<<blocksPerGrid, threadsPerBlock,0, stream1>>>(d_sig_doppler_wiped, d_sig_in,rem_carrier_phase_in_rad,phase_step_rad, signal_length_samples);


    //printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    //wait for Doppler wipeoff end...
    checkCudaErrors(hipStreamSynchronize(stream1));
    checkCudaErrors(hipStreamSynchronize(stream2));
    //checkCudaErrors(hipDeviceSynchronize());

    //old
//    scalarProdGPUCPXxN<<<blocksPerGrid, threadsPerBlock,0 ,stream2>>>(
//    		d_corr_out,
//    		d_sig_doppler_wiped,
//    		d_local_codes_in,
//            3,
//            signal_length_samples
//        );

    //new
    //launch the multitap correlator
    scalarProdGPUCPXxN_shifts<<<blocksPerGrid, threadsPerBlock,0 ,stream2>>>(
			d_corr_out,
			d_sig_doppler_wiped,
			d_local_codes_in,
			d_shifts_samples,
			n_correlators,
			signal_length_samples
		);
    checkCudaErrors(hipGetLastError());
    //wait for correlators end...
    checkCudaErrors(hipStreamSynchronize(stream2));
    // Copy the device result vector in device memory to the host result vector
    // in host memory.

    //scalar products (correlators outputs)
    checkCudaErrors(hipMemcpy(corr_out, d_corr_out, sizeof(std::complex<float>)*n_correlators,
            hipMemcpyDeviceToHost));
    return true;
}

bool cuda_multicorrelator::free_cuda()
{
    hipSetDevice(selected_device); //generates a random number between 0 and num_devices to split the threads between GPUs
	// Free device global memory
	hipFree(d_sig_in);
	//hipFree(d_nco_in);
	hipFree(d_local_codes_in);
	hipFree(d_corr_out);

	hipStreamDestroy(stream1) ;
	hipStreamDestroy(stream2) ;

    // Reset the device and exit
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
	//checkCudaErrors(hipDeviceReset());
	return true;
}

